#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define THREADS 256

// ================= GPU Kernel =================
__global__ void bfs_gpu_kernel(
    int *d_row_ptr,
    int *d_col_idx,
    int *d_frontier,
    int *d_next_frontier,
    int *d_visited,
    int *d_level,
    int n,
    int depth
) {
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if (u >= n) return;
    if (!d_frontier[u]) return;

    for (int e = d_row_ptr[u]; e < d_row_ptr[u+1]; e++) {
        int v = d_col_idx[e];
        if (atomicCAS(&d_visited[v], 0, 1) == 0) {
            d_level[v] = depth + 1;
            d_next_frontier[v] = 1;
        }
    }
}

// ================= CSR Builder =================
void buildCSR(int n, int m, int (*edges)[2], int *row_ptr, int *col_idx) {
    int *deg = (int*)calloc(n, sizeof(int));
    for (int i = 0; i < m; i++) {
        int u = edges[i][0], v = edges[i][1];
        deg[u]++;
        deg[v]++; // undirected
    }

    row_ptr[0] = 0;
    for (int i = 0; i < n; i++) {
        row_ptr[i+1] = row_ptr[i] + deg[i];
    }

    int *temp = (int*)calloc(n, sizeof(int));
    for (int i = 0; i < n; i++) temp[i] = row_ptr[i];

    for (int i = 0; i < m; i++) {
        int u = edges[i][0], v = edges[i][1];
        col_idx[temp[u]++] = v;
        col_idx[temp[v]++] = u;
    }

    free(temp);
    free(deg);
}

// ================= Main =================
int main(int argc, char *argv[]) {
    if (argc < 4) {
        printf("Usage: %s <graph.txt> <partition.csv> <source>\n", argv[0]);
        return 1;
    }

    char *graphFile = argv[1];
    char *partFile  = argv[2];
    int source      = atoi(argv[3]);

    // ---- Read graph ----
    FILE *fin = fopen(graphFile, "r");
    if (!fin) { printf("Error opening %s\n", graphFile); return 1; }

    int n, m;
    fscanf(fin, "%d %d", &n, &m);
    int (*edges)[2] = (int(*)[2]) malloc(m * sizeof(int[2]));
    for (int i = 0; i < m; i++) fscanf(fin, "%d %d", &edges[i][0], &edges[i][1]);
    fclose(fin);

    // ---- Read partition ----
    int *isGPU = (int*)calloc(n, sizeof(int));
    int *isCPU = (int*)calloc(n, sizeof(int));

    FILE *pin = fopen(partFile, "r");
    if (!pin) { printf("Error opening %s\n", partFile); return 1; }
    char line[128];
    fgets(line, sizeof(line), pin); // skip header
    while (fgets(line, sizeof(line), pin)) {
        int vid;
        char part[16];
        sscanf(line, "%d,%15s", &vid, part);
        if (strcmp(part, "GPU") == 0) isGPU[vid] = 1;
        else isCPU[vid] = 1;
    }
    fclose(pin);

    // ---- Build CSR ----
    int *row_ptr = (int*)malloc((n+1)*sizeof(int));
    int *col_idx = (int*)malloc(2*m*sizeof(int));
    buildCSR(n, m, edges, row_ptr, col_idx);
    free(edges);

    // ---- Host arrays ----
    int *frontier      = (int*)calloc(n, sizeof(int));
    int *next_frontier = (int*)calloc(n, sizeof(int));
    int *visited       = (int*)calloc(n, sizeof(int));
    int *level         = (int*)malloc(n*sizeof(int));
    for (int i = 0; i < n; i++) level[i] = -1;

    frontier[source] = 1;
    visited[source] = 1;
    level[source]   = 0;

    // ---- Device arrays ----
    int *d_row_ptr, *d_col_idx, *d_frontier, *d_next_frontier, *d_visited, *d_level;
    hipMalloc(&d_row_ptr, (n+1)*sizeof(int));
    hipMalloc(&d_col_idx, (2*m)*sizeof(int));
    hipMalloc(&d_frontier, n*sizeof(int));
    hipMalloc(&d_next_frontier, n*sizeof(int));
    hipMalloc(&d_visited, n*sizeof(int));
    hipMalloc(&d_level, n*sizeof(int));

    hipMemcpy(d_row_ptr, row_ptr, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx, (2*m)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_visited, visited, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_level, level, n*sizeof(int), hipMemcpyHostToDevice);

    // ---- BFS Loop ----

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    int depth = 0;
    while (1) {
        memset(next_frontier, 0, n*sizeof(int));
        hipMemset(d_next_frontier, 0, n*sizeof(int));

        hipMemcpy(d_frontier, frontier, n*sizeof(int), hipMemcpyHostToDevice);

        // GPU phase
        int blocks = (n + THREADS - 1) / THREADS;
        bfs_gpu_kernel<<<blocks, THREADS>>>(d_row_ptr, d_col_idx,
                                            d_frontier, d_next_frontier,
                                            d_visited, d_level, n, depth);


        // CPU phase
        #pragma omp parallel for schedule(dynamic,64)
        for (int u = 0; u < n; u++) {
            if (!isCPU[u] || !frontier[u]) continue;
            for (int e = row_ptr[u]; e < row_ptr[u+1]; e++) {
                int v = col_idx[e];
                if (!visited[v]) {


                            visited[v] = 1;
                            level[v] = depth+1;
                            next_frontier[v] = 1;
                }
            }
        }
                hipDeviceSynchronize();

        // Merge GPU results
        int *gpu_next = (int*)malloc(n*sizeof(int));
        hipMemcpy(gpu_next, d_next_frontier, n*sizeof(int), hipMemcpyDeviceToHost);

        int any_set = 0;


        for (int i = 0; i < n; i++) {
            frontier[i] = next_frontier[i] || gpu_next[i];
            if (frontier[i]) { visited[i] = 1; any_set = 1; }
        }
        free(gpu_next);

        if (!any_set) break;
        depth++;
    }

        hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("BFS Time: %f ms\n", milliseconds);  

    // ---- Save BFS output ----
    FILE *fout = fopen("bfs_output.csv", "w");
    fprintf(fout, "vertex,level\n");
    for (int i = 0; i < n; i++) {
        fprintf(fout, "%d,%d\n", i, level[i]);
    }
    fclose(fout);

    printf("BFS completed. Results saved to bfs_output.csv\n");

    // ---- Cleanup ----
    free(row_ptr); free(col_idx);
    free(frontier); free(next_frontier);
    free(visited); free(level);
    free(isGPU); free(isCPU);
    hipFree(d_row_ptr); hipFree(d_col_idx);
    hipFree(d_frontier); hipFree(d_next_frontier);
    hipFree(d_visited); hipFree(d_level);

    return 0;
}
